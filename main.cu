#include "hip/hip_runtime.h"
#include <gtk/gtk.h>
#include <gdk/gdkkeysyms.h>
#include <iostream>
#include <sstream>
#include <iomanip>
#include <complex>
#include <ctime>
#include <fstream>
#include <vector>
#include <algorithm>
#include "Common.h"
#include "Timer.h"
#include "WindowInit.h"
#include "kernel.cuh"
using namespace std;

// http://developer.download.nvidia.com/compute/cuda/4_1/rel/toolkit/docs/online/group__CUDART__DEVICE_g028e5b0474379eaf5f5d54657d48600b.html#g028e5b0474379eaf5f5d54657d48600b

#define CPU	0
//#define PIXEL_PER_THREAD

GtkWidget *window;
GtkWidget *da;
GtkWidget *statusBar;
GdkPixbuf *pixbuf;

byte *rawBuffer;
byte *deviceBuffer;

byte *currentPalette;
byte *devicePalette;

int currentPaletteID = 0;

int blockWidth = 16;
int blockHeight = 16;
int threads = 100000;

int devicesCount, currentDevice = 0;
hipDeviceProp_t *deviceProps;

float centerX, centerY, scale;
int iterations = 512;

int bufferWidth = 640;
int bufferHeight = 480;

int viewportWidth, viewportHeight;
int supersampling = 1;

int lastCanvasWidth = 0;
int lastCanvasHeight = 0;

bool initDone = false;

// TEMP
double globalTime;

void updateStatusBar(double time);

void setDefaultView()
{
	centerX = -0.7f;
	centerY = 0.0f;
	scale = 3.0f;
}

static gboolean draw_cb(GtkWidget *widget, cairo_t *cr, gpointer data)
{   
	cairo_scale(cr, 1.0/(double)supersampling, 1.0/(double)supersampling);
	gdk_cairo_set_source_pixbuf(cr, pixbuf, 0, 0);
	cairo_paint(cr);

	return FALSE;
}

void updateBuffer()
{
	int bufferPos = 0;
	
	Timer timer;
	timer.start();

	double ratio = (double)bufferWidth/(double)bufferHeight;
	
	if (currentDevice == CPU)
	{
		for (int y=0; y<bufferHeight; y++)
		{
			for (int x=0; x<bufferWidth; x++)
			{		
				complex<double> c((double)(x - bufferWidth/2)*scale*ratio/(double)(bufferWidth - 1) + centerX,
					(double)(y - bufferHeight/2)*scale/(double)(bufferHeight - 1) + centerY);
				complex<double> z(0.0, 0.0);
				int i = 510;
			
				// checking if we're in the cardioid
				//double q = (real(c) - 0.25)*(real(c) - 0.25) + imag(c)*imag(c);
			
				//if ((q*(q + (real(c) - 0.25)) >= 0.25*imag(c)*imag(c)) && ((real(c) + 1)*(real(c) + 1) + imag(c)*imag(c) >= 0.0625))
				{
					for (i=0; i<iterations; i++)
					{
						z = z*z + c;
					
						if (real(z)*real(z) + imag(z)*imag(z) > 4.0)
							break;
					}
				}
	
				rawBuffer[bufferPos++] = currentPalette[i*3];
				rawBuffer[bufferPos++] = currentPalette[i*3 + 1];
				rawBuffer[bufferPos++] = currentPalette[i*3 + 2];
			}
		}
	}
	else
	{
#ifdef PIXEL_PER_THREAD
		dim3 threads(blockWidth, blockHeight);
		dim3 grid((bufferWidth + (blockWidth - 1))/blockWidth, (bufferHeight + (blockHeight - 1))/blockHeight);

		mandelbrotPixel<<<grid, threads>>>(deviceBuffer, devicePalette, bufferWidth, bufferHeight, centerX, centerY, scale, iterations);
#else

		mandelbrotThread<<<(bufferWidth*bufferHeight + blockWidth - 1)/blockWidth, blockWidth>>>
			(deviceBuffer, devicePalette, bufferWidth, bufferHeight, threads, centerX, centerY, scale, iterations);
#endif
		hipError_t err = hipSuccess; 
		err = hipGetLastError();
		cerr << "Failed to launch kernel (error code %s)! " << hipGetErrorString(err) << endl;
		
		hipMemcpy(rawBuffer, deviceBuffer, bufferWidth*bufferHeight*3, hipMemcpyDeviceToHost);
	}
	
	timer.stop();
	gtk_widget_queue_draw(da);
	updateStatusBar(timer.getElapsedTimeInSec());
}

void updateStatusBar(double time)
{
	globalTime = time;

	ostringstream newStatus;
	newStatus << fixed << setprecision(5) << "Center: " << centerX << " " << showpos << centerY << "i   Scale: "
		<< noshowpos << scale << "   Iterations: " << iterations << "   Buffer: " << bufferWidth << "x" << bufferHeight << "   Time: " << time << " s   |   ";
	
	if (currentDevice == 0)
		newStatus << "CPU";
	else
	{
		int memInMB = (deviceProps[currentDevice - 1].totalGlobalMem + 1024*1024 - 1)/(1024*1024);
		newStatus << deviceProps[currentDevice - 1].name << "    " << memInMB << " MB    CUDA Compute Capability: " <<
			deviceProps[currentDevice - 1].major << "." << deviceProps[currentDevice - 1].minor;
	}
		
	gtk_statusbar_push(GTK_STATUSBAR(statusBar), 0, newStatus.str().c_str());
}

void reallocateFrameBuffer()
{
	bufferWidth = viewportWidth*supersampling;
	bufferHeight = viewportHeight*supersampling;

	// reallocating local frame buffer
	delete[] rawBuffer;
		
	if (pixbuf != NULL)
		g_object_unref(pixbuf);

	rawBuffer = new byte[bufferWidth*bufferHeight*3];
	pixbuf = gdk_pixbuf_new_from_data(rawBuffer, GDK_COLORSPACE_RGB,
		FALSE, 8, bufferWidth, bufferHeight, bufferWidth*3, NULL, NULL);

	// (re)allocating CUDA device's frame buffer

	if (currentDevice != CPU)
	{
		if (deviceBuffer != 0)
			hipFree(deviceBuffer);
			
		hipMalloc((void**)&deviceBuffer, bufferWidth*bufferHeight*3);
	}
}

gboolean canvasFrameChanged(GtkWindow *window, GdkEvent *event, gpointer data)
{
	if (lastCanvasWidth != event->configure.width ||
		lastCanvasHeight != event->configure.height)
	{
		viewportWidth = event->configure.width;
		viewportHeight = event->configure.height;

		reallocateFrameBuffer();
		updateBuffer();
	
		lastCanvasWidth = event->configure.width;
		lastCanvasHeight = event->configure.height;
	}
	
	return FALSE;
}

void menuitem_response(GtkWidget *widget, int device)
{
	if (device != currentDevice)
	{
		// changing the current device
		currentDevice = device;
	
		if (device != CPU)
		{
			hipSetDevice(device - 1);
		
			if (deviceBuffer != 0)
				hipFree(deviceBuffer);
				
			if (devicePalette != NULL)
				hipFree(devicePalette);
				
			hipMalloc((void**)&deviceBuffer, bufferWidth*bufferHeight*3);

			hipMalloc((void**)&devicePalette, (iterations + 2)*3);
			hipMemcpy(devicePalette, currentPalette, (iterations + 2)*3, hipMemcpyHostToDevice);
		}

		updateBuffer();
	}
}

void create_dialog(GtkWindow *window, char *title, char *message)
{
    GtkWidget *dialog, *label, *content_area;

    /* New label for dialog content */
    label = gtk_label_new(message);

    /* Make a new dialog with an 'OK' button */
    dialog = gtk_dialog_new_with_buttons(title, window, GTK_DIALOG_DESTROY_WITH_PARENT, GTK_STOCK_OK, GTK_RESPONSE_NONE, NULL);

    /* Add label to dialog */
    content_area = gtk_dialog_get_content_area(GTK_DIALOG(dialog));
    gtk_container_add(GTK_CONTAINER(content_area), label);

    /* Destroy dialog properly */
    g_signal_connect(dialog, "response", G_CALLBACK(gtk_widget_destroy), dialog);

    /* Set dialog to not resize. */
    gtk_window_set_resizable(GTK_WINDOW(dialog), FALSE);

    gtk_widget_show_all(dialog);
}

void openHelp(GtkWidget *widget, int whichWindow)
{
	ofstream outfile;
	outfile.open("measurements.txt", std::ios_base::out);

	for(int y=1; y<=40; y++)
	{
		blockHeight = y;

		for(int x=1; x<=40; x++)
		{
			if (x*y > 512)
			{
				outfile << "\t";
				continue;
			}

			blockWidth = x;

			double sum = 0.0;

			for (int i=0; i<50; i++)
			{
				updateBuffer();
				sum += globalTime;
			}

			double avgTime = sum/50.0;

			outfile << avgTime << "\t";
			cerr << x << ", " << y << " -> " << avgTime << endl;
		}

		outfile << endl;
	}


	if (whichWindow == 0)
		create_dialog((GtkWindow*)window, "Usage", "Arrow keys: moving the view up/down and left/right\nPlus\\minus keys: zooming in\\out\n\nQ\\A: increasing\\decreasing the number of iterations\nW\\S: increasing\\decreasing the number of iterations by 100");
	else
		create_dialog((GtkWindow*)window, "About", "Mandelbrot Explorer\nby Piotr Krzeminski, 131546\n\nThis application has been created as a project\nfor \"CUDA\\CELL processing\" university course.");
}

int amplify(int val)
{
	float floatVal = (float)val/255.0f;
	float amplified = sqrtf(floatVal);
	
	return (int)(amplified*255.0f);
}

void generatePalette(int paletteID)
{
	if (currentPalette != NULL)
		delete[] currentPalette;

	currentPalette = new byte[(iterations + 2)*3];

	int arrayPos = 0;

	// grayscale
	if (paletteID == 0)
	{
		for (int i=0; i<=iterations; i++)
		{
			currentPalette[arrayPos++] = 255 - amplify(i*255/iterations);
			currentPalette[arrayPos++] = 255 - amplify(i*255/iterations);
			currentPalette[arrayPos++] = 255 - amplify(i*255/iterations);
		}
	}
	else if (paletteID == 1)
	{
		for (int i=0; i<iterations; i++)
		{
			currentPalette[arrayPos++] = amplify(i <= iterations/2 ? 0 : (i - iterations/2)*2*255/iterations);
			currentPalette[arrayPos++] = amplify(i <= iterations/2 ? i*2*255/iterations : 255);
			currentPalette[arrayPos++] = amplify(i <= iterations/2 ? 0 : (i - iterations/2)*2*255/iterations);
		}
			
		currentPalette[arrayPos++] = 0;
		currentPalette[arrayPos++] = 0;
		currentPalette[arrayPos++] = 0;			
	}
	else if (paletteID == 2)
	{
		for (int i=0; i<iterations; i++)
		{
			currentPalette[arrayPos++] = 255 - (i&1)*255;
			currentPalette[arrayPos++] = 255 - (i&1)*255;
			currentPalette[arrayPos++] = 255 - (i&1)*255;
		}

		currentPalette[arrayPos++] = 128;
		currentPalette[arrayPos++] = 128;
		currentPalette[arrayPos++] = 128;		
	}

	if (currentDevice != CPU)
	{
		if (devicePalette != NULL)
			hipFree(devicePalette);

		hipMalloc((void**)&devicePalette, (iterations + 2)*3);
		hipMemcpy(devicePalette, currentPalette, (iterations + 2)*3, hipMemcpyHostToDevice);
	}

	currentPaletteID = paletteID;
}

void paletteChanged(GtkWidget *widget, int paletteID)
{
	if (paletteID != currentPaletteID)
	{
		generatePalette(paletteID);
		updateBuffer();
	}
}

void antialiasingChanged(GtkWidget *widget, int aaID)
{
	if (aaID != supersampling)
	{
		supersampling = aaID;
		reallocateFrameBuffer();
		updateBuffer();
	}
}

void blockSizeChanged(GtkWidget *widget, int blockSize)
{
	if (blockWidth != blockSize && initDone == true)
	{
		blockWidth = blockHeight = blockSize;
		updateBuffer();
	}
}

gboolean onKeyPress(GtkWidget *widget, GdkEventKey *event, gpointer user_data)
{
	switch (event->keyval)
	{
		case GDK_KEY_Left:
			centerX -= 0.05f*scale;
			break;
		case GDK_KEY_Right:
			centerX += 0.05f*scale;
			break;
		case GDK_KEY_Up:
			centerY -= 0.05f*scale;
			break;
		case GDK_KEY_Down:
			centerY += 0.05f*scale;
			break;
		case GDK_KEY_equal:
		case GDK_KEY_KP_Add:
			scale /= 1.1f;
			break;
		case GDK_KEY_minus:
		case GDK_KEY_KP_Subtract:
			scale *= 1.1f;
			break;
		case GDK_KEY_r:
			setDefaultView();
			break;
		case GDK_KEY_q:
			iterations++;
			generatePalette(currentPaletteID);
			break;
		case GDK_KEY_a:
			iterations = max(1, iterations - 1);
			generatePalette(currentPaletteID);
			break;
		case GDK_KEY_w:
			iterations += 100;
			generatePalette(currentPaletteID);
			break;
		case GDK_KEY_s:
			iterations = max(1, iterations - 100);
			generatePalette(currentPaletteID);
			break;
		default:
			return FALSE;
	}
	
	updateBuffer();
	gtk_widget_queue_draw(da);
	
	return FALSE;
}

void initWindow()
{
	// creating a window
	window = createWindow(G_CALLBACK(onKeyPress));
	
	// creating a main menu
	GtkWidget *menuBar = gtk_menu_bar_new();

	// adding subsequent submenus
	gtk_menu_shell_append(GTK_MENU_SHELL(menuBar), createDeviceMenu(G_CALLBACK(menuitem_response)));
	gtk_menu_shell_append(GTK_MENU_SHELL(menuBar), createPaletteMenu(G_CALLBACK(paletteChanged)));
	gtk_menu_shell_append(GTK_MENU_SHELL(menuBar), createAntialiasingMenu(G_CALLBACK(antialiasingChanged)));
	gtk_menu_shell_append(GTK_MENU_SHELL(menuBar), createBlockSizeMenu(G_CALLBACK(blockSizeChanged)));
	gtk_menu_shell_append(GTK_MENU_SHELL(menuBar), createHelpMenu(G_CALLBACK(openHelp)));
	
	// creating a drawing area
	da = createDrawingArea(G_CALLBACK(draw_cb), G_CALLBACK(canvasFrameChanged));
	
	// creating a status bar
	statusBar = gtk_statusbar_new();

	// adding all the elements to the window, stacked vertically
	GtkWidget *vbox = gtk_box_new(GTK_ORIENTATION_VERTICAL, 0);
	gtk_container_add(GTK_CONTAINER(window), vbox);
	gtk_box_pack_start(GTK_BOX(vbox), menuBar, FALSE, FALSE, 0);
	gtk_box_pack_start(GTK_BOX(vbox), da, TRUE, TRUE, 0);
	gtk_box_pack_start(GTK_BOX(vbox), statusBar, FALSE, FALSE, 3);
	
	// displaying the window
	gtk_widget_show_all(window);
}

int main(int argc, char *argv[])
{
	gtk_init(&argc, &argv);

	// setting initial values
	setDefaultView();
	generatePalette(1);
	
	initWindow();
	initDone = true;

	// the main loop
	gtk_main();

	return 0;
}
