#include "hip/hip_runtime.h"
#include <gtk/gtk.h>
#include <gdk/gdkkeysyms.h>
#include <iostream>
#include <sstream>
#include <iomanip>
#include <complex>
#include <ctime>
#include <hip/hip_runtime.h>
#include "Timer.h"
using namespace std;
typedef unsigned char byte;

// http://developer.download.nvidia.com/compute/cuda/4_1/rel/toolkit/docs/online/group__CUDART__DEVICE_g028e5b0474379eaf5f5d54657d48600b.html#g028e5b0474379eaf5f5d54657d48600b

#define WINDOW_WIDTH  1024
#define WINDOW_HEIGHT 768

#define CPU	0

GtkWidget *da;
GtkWidget *statusBar;
GdkPixbuf *pixbuf;

byte *rawBuffer;
byte *deviceBuffer;

byte *currentPalette;
byte *devicePalette;

int currentPaletteID = 0;

int devicesCount, currentDevice = 0;
hipDeviceProp_t *deviceProps;

double centerX, centerY, scale;

int bufferWidth = 640;
int bufferHeight = 480;

int lastCanvasWidth = 0;
int lastCanvasHeight = 0;

void updateStatusBar(double time);

void setDefaultView()
{
	centerX = -0.7;
	centerY = 0.0;
	scale = 3.0;
}

static gboolean draw_cb(GtkWidget *widget, cairo_t *cr, gpointer data)
{   
	gdk_cairo_set_source_pixbuf(cr, pixbuf, 0, 0);
	cairo_paint(cr);
	cairo_fill(cr);

	return FALSE;
}

__global__ void mandelbrotPixel(byte *output, byte *palette, int width, int height, double centerX, double centerY, double scale)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    
    if ((x >= width) || (y >= height))
    	return;
    	
    double cReal, cImag;
    cReal = (double)(x - width/2)*scale/(double)(width - 1) + centerX;
    cImag = (double)(y - height/2)*scale/(double)(height - 1) + centerY;
    
	double zReal = 0.0f, zImag = 0.0f, z2Real, z2Imag;
	
	int i;
	
	for (i = 0; i<510; i++)
	{
		z2Real = zReal*zReal - zImag*zImag + cReal;
		z2Imag = 2.0f*zReal*zImag + cImag;
		
		zReal = z2Real;
		zImag = z2Imag;
		
		if (zReal*zReal + zImag*zImag > 4.0f)
			break;
	}
		
	int bufferPos = (width*y + x)*3;
		
	output[bufferPos++] = palette[i*3];
	output[bufferPos++] = palette[i*3 + 1];
	output[bufferPos++] = palette[i*3 + 2];
}

void updateBuffer()
{
	int bufferPos = 0;
	
	Timer timer;
	timer.start();
	
	if (currentDevice == CPU)
	{
		for (int y=0; y<bufferHeight; y++)
		{
			for (int x=0; x<bufferWidth; x++)
			{		
				complex<double> c((double)(x - bufferWidth/2)*scale/(double)(bufferWidth - 1) + centerX,
					(double)(y - bufferHeight/2)*scale/(double)(bufferHeight - 1) + centerY);
				complex<double> z(0.0, 0.0);
				int i = 510;
			
				// checking if we're in the cardioid
				double q = (real(c) - 0.25)*(real(c) - 0.25) + imag(c)*imag(c);
			
				if ((q*(q + (real(c) - 0.25)) >= 0.25*imag(c)*imag(c)) && ((real(c) + 1)*(real(c) + 1) + imag(c)*imag(c) >= 0.0625))
				{
					for (i=0; i<510; i++)
					{
						z = z*z + c;
					
						if (real(z)*real(z) + imag(z)*imag(z) > 4.0)
							break;
					}
				}
	
				rawBuffer[bufferPos++] = currentPalette[i*3];
				rawBuffer[bufferPos++] = currentPalette[i*3 + 1];
				rawBuffer[bufferPos++] = currentPalette[i*3 + 2];
			}
		}
	}
	else
	{
		if (deviceBuffer != 0)
			hipFree(deviceBuffer);
			
		hipMalloc((void**)&deviceBuffer, bufferWidth*bufferHeight*3);
		hipMalloc((void**)&devicePalette, 512*3);
		
		hipMemcpy(devicePalette, currentPalette, 512*3, hipMemcpyHostToDevice);
		
		dim3 threads(8, 8);
		dim3 grid((bufferWidth + 7)/8, (bufferHeight + 7)/8);
	
		mandelbrotPixel<<<grid, threads>>>(deviceBuffer, devicePalette, bufferWidth, bufferHeight, centerX, centerY, scale);
		
		//hipError_t err = hipSuccess; 
		//err = hipGetLastError();
		//cerr << "Failed to launch kernel (error code %s)! " << hipGetErrorString(err);
		
		hipMemcpy(rawBuffer, deviceBuffer, bufferWidth*bufferHeight*3, hipMemcpyDeviceToHost);
	}
	
	timer.stop();
	updateStatusBar(timer.getElapsedTimeInSec());
}

void updateStatusBar(double time)
{
	ostringstream newStatus;
	newStatus << fixed << setprecision(5) << "Center: " << centerX << " " << showpos << centerY << "i   Scale: "
		<< noshowpos << scale << "   Time: " << time << " ms   |   ";
	
	if (currentDevice == 0)
		newStatus << "CPU";
	else
	{
		int memInMB = (deviceProps[currentDevice - 1].totalGlobalMem + 1024*1024 - 1)/(1024*1024);
		newStatus << deviceProps[currentDevice - 1].name << "    " << memInMB << " MB";
	}
		
	gtk_statusbar_push(GTK_STATUSBAR(statusBar), 0, newStatus.str().c_str());
	gtk_widget_queue_draw(statusBar);
}

gboolean canvasFrameChanged(GtkWindow *window, GdkEvent *event, gpointer data)
{
	if (lastCanvasWidth != event->configure.width ||
		lastCanvasHeight != event->configure.height)
	{
		delete[] rawBuffer;
		
		if (pixbuf != NULL)
			g_object_unref(pixbuf);
		
		bufferWidth = event->configure.width;
		bufferHeight = event->configure.height;
		
		rawBuffer = new byte[bufferWidth*bufferHeight*3];
		pixbuf = gdk_pixbuf_new_from_data(rawBuffer, GDK_COLORSPACE_RGB,
			FALSE, 8, bufferWidth, bufferHeight, bufferWidth*3, NULL, NULL);			

		updateBuffer();
	
		lastCanvasWidth = event->configure.width;
		lastCanvasHeight = event->configure.height;
	}
	
	return FALSE;
}

void menuitem_response(GtkWidget *widget, int device)
{
	if (device != currentDevice)
	{
		// changing the current device
		currentDevice = device;
	
		if (device != CPU)
		{
			hipSetDevice(device - 1);
		}

		updateBuffer();
	}
}

int amplify(int val)
{
	float floatVal = (float)val/255.0f;
	float amplified = sqrtf(floatVal);
	
	return (int)(amplified*255.0f);
}

void paletteChanged(GtkWidget *widget, int paletteID)
{
	if (paletteID != currentPaletteID)
	{
		int arrayPos = 0;
		
		// grayscale
		if (paletteID == 0)
		{
			for (int i=0; i<511; i++)
			{
				currentPalette[arrayPos++] = 255 - amplify(i/2);
				currentPalette[arrayPos++] = 255 - amplify(i/2);
				currentPalette[arrayPos++] = 255 - amplify(i/2);
			}
		}
		else if (paletteID == 1)
		{
			for (int i=0; i<510; i++)
			{
				currentPalette[arrayPos++] = amplify(i <= 255 ? 0 : i - 256);
				currentPalette[arrayPos++] = amplify(i <= 255 ? i : 255);
				currentPalette[arrayPos++] = amplify(i <= 255 ? 0 : i - 256);
			}
			
			currentPalette[arrayPos++] = 0;
			currentPalette[arrayPos++] = 0;
			currentPalette[arrayPos++] = 0;			
		}
		
		currentPaletteID = paletteID;
		updateBuffer();
	}
}

gboolean onKeyPress(GtkWidget *widget, GdkEventKey *event, gpointer user_data)
{
	switch (event->keyval)
	{
		case GDK_KEY_Left:
			centerX -= 0.05*scale;
			break;
		case GDK_KEY_Right:
			centerX += 0.05*scale;
			break;
		case GDK_KEY_Up:
			centerY -= 0.05*scale;
			break;
		case GDK_KEY_Down:
			centerY += 0.05*scale;
			break;
		case GDK_KEY_equal:
		case GDK_KEY_KP_Add:
			scale /= 1.1;
			break;
		case GDK_KEY_minus:
		case GDK_KEY_KP_Subtract:
			scale *= 1.1;
			break;
		case GDK_KEY_r:
			setDefaultView();
			break;
		default:
			return FALSE;
	}
	
	updateBuffer();

	gtk_widget_queue_draw(da);
	
	return FALSE;
}

int main(int argc, char *argv[])
{
	setDefaultView();
	
	// ----------------

	currentPalette = new byte[512*3];
	int arrayPos = 0;
	
	for (int i=0; i<510; i++)
	{
		currentPalette[arrayPos++] = amplify(i <= 255 ? 0 : i - 256);
		currentPalette[arrayPos++] = amplify(i <= 255 ? i : 255);
		currentPalette[arrayPos++] = amplify(i <= 255 ? 0 : i - 256);
	}
	
	currentPalette[arrayPos++] = 0;
	currentPalette[arrayPos++] = 0;
	currentPalette[arrayPos++] = 0;		
	
	// ----------------

	gtk_init(&argc, &argv);
	
	// creating a window
	GtkWidget *window = gtk_window_new(GTK_WINDOW_TOPLEVEL);
	gtk_window_set_default_size((GtkWindow*)window, WINDOW_WIDTH, WINDOW_HEIGHT);
	gtk_window_set_title((GtkWindow*)window, "Mandelbrot Explorer");
	g_signal_connect(window, "destroy", G_CALLBACK(gtk_main_quit), NULL);
	g_signal_connect(window, "key_press_event", G_CALLBACK(onKeyPress), NULL);
	
	// creating the main menu
	GtkWidget *vbox = gtk_box_new(GTK_ORIENTATION_VERTICAL, 0);
	gtk_container_add(GTK_CONTAINER(window), vbox);
	
	GtkWidget *menuBar = gtk_menu_bar_new();
	GtkWidget *deviceMenu = gtk_menu_new();
	
	// "Device" menu and radio buttons
	GtkWidget *deviceMenuItem = gtk_menu_item_new_with_label("Device");
	GSList *devicesRadioGroup = NULL;
	
	GtkWidget *cpuMenuItem = gtk_radio_menu_item_new_with_label(devicesRadioGroup, "CPU");
	gtk_menu_shell_append(GTK_MENU_SHELL(deviceMenu), cpuMenuItem);
	g_signal_connect(cpuMenuItem, "activate", G_CALLBACK(menuitem_response), (gpointer)0);
	devicesRadioGroup = gtk_radio_menu_item_get_group(GTK_RADIO_MENU_ITEM(cpuMenuItem));
	gtk_check_menu_item_set_active(GTK_CHECK_MENU_ITEM(cpuMenuItem), TRUE);
	
	// getting info about installed CUDA-capable devices
	
	hipGetDeviceCount(&devicesCount);
	deviceProps = new hipDeviceProp_t[devicesCount];
	
	for (int i=0; i<devicesCount; i++)
	{
		hipGetDeviceProperties(&deviceProps[i], i);

		GtkWidget *deviceMenuItem = gtk_radio_menu_item_new_with_label(devicesRadioGroup, deviceProps[i].name);
		gtk_menu_shell_append(GTK_MENU_SHELL(deviceMenu), deviceMenuItem);
		g_signal_connect(deviceMenuItem, "activate", G_CALLBACK(menuitem_response), (gpointer)(i + 1));
				
		devicesRadioGroup = gtk_radio_menu_item_get_group(GTK_RADIO_MENU_ITEM(deviceMenuItem));
	}
	
	gtk_menu_item_set_submenu(GTK_MENU_ITEM(deviceMenuItem), deviceMenu);
	gtk_menu_shell_append(GTK_MENU_SHELL(menuBar), deviceMenuItem);
	
	// "Palette" menu and radio buttons
	GtkWidget *paletteMenu = gtk_menu_new();
	GtkWidget *paletteMenuItem = gtk_menu_item_new_with_label("Palette");
	GSList *palettesRadioGroup = NULL;
	GtkWidget *grayscaleMenuItem = gtk_radio_menu_item_new_with_label(palettesRadioGroup, "Grayscale");
	
	palettesRadioGroup = gtk_radio_menu_item_get_group(GTK_RADIO_MENU_ITEM(grayscaleMenuItem));
	GtkWidget *blackGreenWhiteMenuItem = gtk_radio_menu_item_new_with_label(palettesRadioGroup, "Black-green-white");
	
	// set "Black-green-white" as currently selected
	gtk_check_menu_item_set_active(GTK_CHECK_MENU_ITEM(blackGreenWhiteMenuItem), TRUE);
	
	gtk_menu_item_set_submenu(GTK_MENU_ITEM(paletteMenuItem), paletteMenu);
	gtk_menu_shell_append(GTK_MENU_SHELL(paletteMenu), grayscaleMenuItem);
	g_signal_connect(grayscaleMenuItem, "activate", G_CALLBACK(paletteChanged), (gpointer)0);
	gtk_menu_shell_append(GTK_MENU_SHELL(paletteMenu), blackGreenWhiteMenuItem);
	g_signal_connect(blackGreenWhiteMenuItem, "activate", G_CALLBACK(paletteChanged), (gpointer)1);
	gtk_menu_shell_append(GTK_MENU_SHELL(menuBar), paletteMenuItem);
	
	// "Help" menu
	GtkWidget *helpMenu = gtk_menu_new();
	GtkWidget *helpMenuItem = gtk_menu_item_new_with_label("Help");
	GtkWidget *usageMenuItem = gtk_menu_item_new_with_label("Usage");
	GtkWidget *aboutMenuItem = gtk_menu_item_new_with_label("About");
	
	gtk_menu_item_set_submenu(GTK_MENU_ITEM(helpMenuItem), helpMenu);
	gtk_menu_shell_append(GTK_MENU_SHELL(helpMenu), usageMenuItem);
	gtk_menu_shell_append(GTK_MENU_SHELL(helpMenu), aboutMenuItem);
	gtk_menu_shell_append(GTK_MENU_SHELL(menuBar), helpMenuItem);
	
	gtk_box_pack_start(GTK_BOX(vbox), menuBar, FALSE, FALSE, 0);
	
	// creating a drawing area
	da = gtk_drawing_area_new();
	g_signal_connect(da, "draw", G_CALLBACK(draw_cb), NULL);
	g_signal_connect(da, "configure-event", G_CALLBACK(canvasFrameChanged), NULL);
	
	gtk_box_pack_start(GTK_BOX(vbox), da, TRUE, TRUE, 0);

	// creating a status bar
	statusBar = gtk_statusbar_new();
	gtk_box_pack_start(GTK_BOX(vbox), statusBar, FALSE, FALSE, 3);
	
	gtk_widget_show_all(window);

	// the main loop
	gtk_main();

	return 0;
}
