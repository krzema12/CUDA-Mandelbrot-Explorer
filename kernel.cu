#include "hip/hip_runtime.h"
#include "kernel.cuh"

__global__ void mandelbrotPixel(byte *output, byte *palette, int width, int height, float centerX, float centerY, float scale, int iterations)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
    
	if ((x >= width) || (y >= height))
		return;

	float ratio = (float)width/(float)height;
    	
	float cReal, cImag;
	cReal = (float)(x - width/2)*scale*ratio/(float)(width - 1) + centerX;
	cImag = (float)(y - height/2)*scale/(float)(height - 1) + centerY;
    
	float zReal = 0.0f, zImag = 0.0, z2Real, z2Imag;
	
	int i;
	
	for (i = 0; i<iterations; i++)
	{
		z2Real = zReal*zReal - zImag*zImag + cReal;
		z2Imag = 2.0f*zReal*zImag + cImag;
		
		zReal = z2Real;
		zImag = z2Imag;
		
		if (zReal*zReal + zImag*zImag > 4.0f)
			break;
	}

	int bufferPos = (width*y + x)*3;
	int paletteIndex = i*3;
		
	output[bufferPos++] = palette[paletteIndex++];
	output[bufferPos++] = palette[paletteIndex++];
	output[bufferPos++] = palette[paletteIndex++];
}
