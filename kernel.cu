#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <math.h>

//#define BITMASK_OPTIMALIZATION

__global__ void mandelbrotPixel(byte *output, byte *palette, int width, int height, float centerX, float centerY, float scale, int iterations)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
    
	if ((x >= width) || (y >= height))
		return;

	float ratio = (float)width/(float)height;
    	
	float cReal, cImag;
	cReal = (float)(x - width/2)*scale*ratio/(float)(width - 1) + centerX;
	cImag = (float)(y - height/2)*scale/(float)(height - 1) + centerY;
    
	float zReal = 0.0f, zImag = 0.0, z2Real, z2Imag;
	
	int i;

#ifdef BITMASK_OPTIMALIZATION
	int iters = 0;
	int doneMask = 0;
	int itersIsEmptyMask = 0;
#endif

	for (i = 0; i<iterations; i++)
	{
		z2Real = zReal*zReal - zImag*zImag + cReal;
		z2Imag = 2.0f*zReal*zImag + cImag;
		
		zReal = z2Real;
		zImag = z2Imag;

#ifdef BITMASK_OPTIMALIZATION
		doneMask = ~((zReal*zReal + zImag*zImag > 4.0f) - 1);	// time to exit the loop  =>  0xFFFFFFFF
		itersIsEmptyMask = (iters != 0) - 1;	// iters == 0  =>  0xFFFFFFFF
		iters = (doneMask & itersIsEmptyMask & i) | ((~itersIsEmptyMask) & iters);
#else
		if (zReal*zReal + zImag*zImag > 4.0f)
			break;
#endif
	}

#ifdef BITMASK_OPTIMALIZATION
	int paletteIndex = iters*3;
#else
	int paletteIndex = i*3;
#endif

	int bufferPos = (width*y + x)*3;

	output[bufferPos++] = palette[paletteIndex++];
	output[bufferPos++] = palette[paletteIndex++];
	output[bufferPos++] = palette[paletteIndex++];
}

__global__ void mandelbrotThread(byte *output, byte *palette, int width, int height, int threads, float centerX, float centerY, float scale, int iterations)
{
	// the host told us to calculate the Mandelbrot using 'threads' number of threads.
	// We have 'width*height' pixels to compute in total, so one thread will calculate
	// approximately 'width*height/threads' pixels.

	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int eachThread = (width*height + threads - 1)/threads;
	int c = tid*eachThread; // 'c' like current pixel index
	int x, y, i;

	float ratio = (float)width/(float)height;

	for (i=0; i<eachThread; i++, c++)
	{
		y = c/width;
		x = c%width;

		if ((x >= width) || (y >= height))
			return;

		float cReal, cImag;
		cReal = (float)(x - width/2)*scale*ratio/(float)(width - 1) + centerX;
		cImag = (float)(y - height/2)*scale/(float)(height - 1) + centerY;
    
		float zReal = 0.0f, zImag = 0.0, z2Real, z2Imag;
	
		int i;

		for (i = 0; i<iterations; i++)
		{
			z2Real = zReal*zReal - zImag*zImag + cReal;
			z2Imag = 2.0f*zReal*zImag + cImag;
		
			zReal = z2Real;
			zImag = z2Imag;

			if (zReal*zReal + zImag*zImag > 4.0f)
				break;
		}

		int paletteIndex = i*3;
		int bufferPos = (width*y + x)*3;

		output[bufferPos++] = palette[paletteIndex++];
		output[bufferPos++] = palette[paletteIndex++];
		output[bufferPos++] = palette[paletteIndex++];
	}
}
